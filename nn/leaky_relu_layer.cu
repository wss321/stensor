#include "hip/hip_runtime.h"
/**
* Copyright 2021 wss
* Created by wss on 11月,26, 2021
*/
#include "leaky_relu_layer.hpp"
#include "math/math_base_cuda.hpp"

namespace stensor {

namespace nn {

template<typename Dtype>
__global__ void leakyrelu_backward_kernel(const int n, const Dtype *y_grad, const Dtype *x_data, Dtype *x_grad, Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype x=x_data[index];
    x_grad[index] += y_grad[index]*(x > 0 ? Dtype(1.0): alpha);
  }
}

void LeakyReLU::backward_gpu() {
  SharedTensor x(inputs_[0]);
  SharedTensor y(outputs_[0]);
  const float *y_grad = y->const_grad();
  const float *x_data = x->const_data();
  float *x_grad = x->grad();

  leakyrelu_backward_kernel<float><<<GET_BLOCKS(x->size()),
  CUDA_NUM_THREADS>>>(x->size(), y_grad, x_data, x_grad, alpha_);

  inputs_.clear();
}

}//namespace nn

}//namespace stensor